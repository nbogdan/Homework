#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * This software and the information contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a Non-Disclosure Agreement.  Any reproduction or
 * disclosure to any third party without the express written consent of
 * NVIDIA is prohibited.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>

// includes, project
#include "2Dconvolution.h"


////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int);

Matrix AllocateDeviceMatrix(int width, int height);
Matrix AllocateMatrix(int width, int height);
void FreeDeviceMatrix(Matrix* M);
void FreeMatrix(Matrix* M);

void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P);
void ConvolutionOnDeviceShared(const Matrix M, const Matrix N, Matrix P);

////////////////////////////////////////////////////////////////////////////////
// Înmulțirea fără memorie partajată
////////////////////////////////////////////////////////////////////////////////
__global__ void ConvolutionKernel(Matrix M, Matrix N, Matrix P)
{

    float newValue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int i, j;
    

    if(row >= N.height || col >= N.width)
        return;

    for(i = 0; i < KERNEL_SIZE; i++)
        for(j = 0; j < KERNEL_SIZE; j++) {

            if(row + i - (KERNEL_SIZE / 2) >= 0 && col + j - (KERNEL_SIZE / 2) >= 0 &&
               row + i - (KERNEL_SIZE / 2) < N.height && col + j - (KERNEL_SIZE / 2) < N.width )
            {
                newValue += N.elements[(row  + i - KERNEL_SIZE / 2)* P.width + col + j - KERNEL_SIZE / 2] * M.elements[i * KERNEL_SIZE + j];
            }
        }
        
    P.elements[row * P.width + col] = newValue;    
}


////////////////////////////////////////////////////////////////////////////////
// Înmulțirea cu memorie partajată
////////////////////////////////////////////////////////////////////////////////
__global__ void ConvolutionKernelShared(Matrix M, Matrix N, Matrix P)
{
    
    __shared__ float Ns[(BLOCK_SIZE + KERNEL_SIZE - 1)*(BLOCK_SIZE + KERNEL_SIZE - 1)];
    __shared__ float Ms[KERNEL_SIZE * KERNEL_SIZE];

    int memorySize = BLOCK_SIZE + KERNEL_SIZE - 1;

    float newValue = 0;

    /* Randul si coloana aferente blocului */
    int blockRow = blockIdx.y * blockDim.y;
    int blockCol = blockIdx.x * blockDim.x;

    /* Randul si coloana aferente thread-ului */
    int row = blockRow + threadIdx.y;
    int col = blockCol + threadIdx.x;

    /* Pozitie thread in cadrul blocului */
    int id = threadIdx.y * blockDim.x + threadIdx.x;
    int i, j;
    
    /* Pozitie in Ns corespunzatoare thread-ului curent */
    int newRow = blockRow - (KERNEL_SIZE / 2) + (id / memorySize);
    int newCol = blockCol - (KERNEL_SIZE / 2) + (id % memorySize);
    
    /* Daca blocul este complet fiecare thread(256 intr-un bloc) va prelua
    cate o valoare din cele 400 disponibile intr-un bloc. */
    Ns[id] = 0;
       if((newRow >= 0 && newRow < N.height) &&
            (newCol >= 0 && newCol < N.width))
                Ns[id] = N.elements[newRow * N.width + newCol];

    
    /* Thread-urile preiau restul de 144 de valori ramase + matricea de kernel */
    id += BLOCK_SIZE * BLOCK_SIZE;
    newRow = blockRow - (KERNEL_SIZE / 2) + (id / memorySize);
    newCol = blockCol - (KERNEL_SIZE / 2) + (id % memorySize);
    if(id < memorySize * memorySize)
    {
        Ns[id] = 0;
        if((newRow >= 0 && newRow < N.height) &&
            ( newCol >= 0 && newCol < N.width))
                Ns[id] = N.elements[newRow * N.width + newCol];
    }
    else if(id < memorySize * memorySize + KERNEL_SIZE * KERNEL_SIZE)
    {
        Ms[id - memorySize * memorySize] = M.elements[id - memorySize * memorySize];
    }

    __syncthreads();/* In acest moment avem Ms si Ns disponibile. */

    if(row >= N.height || col >= N.width)
        return;
    
    /* Initializam valoarea din matricea rezultat pentru a nu avea surprize. */
    P.elements[row * P.width + col] = 0;
        
    /* Calculam noua valoare a pixel-ului. */
    for(i = 0; i < KERNEL_SIZE; i++)
        for(j = 0; j < KERNEL_SIZE; j++)
            newValue += Ns[(row + i - blockRow) * memorySize + col - blockCol + j] * Ms[i * KERNEL_SIZE + j];  
    
    P.elements[row * P.width + col] = newValue;
}

////////////////////////////////////////////////////////////////////////////////
// Returnează 1 dacă matricele sunt ~ egale
////////////////////////////////////////////////////////////////////////////////
int CompareMatrices(Matrix A, Matrix B)
{
    int i;
    if(A.width != B.width || A.height != B.height || A.pitch != B.pitch)
        return 0;
    int size = A.width * A.height;
    for(i = 0; i < size; i++)
        if(fabs(A.elements[i] - B.elements[i]) > MAX_ERR)
            return 0;
    return 1;
}
void GenerateRandomMatrix(Matrix m)
{
    int i;
    int size = m.width * m.height;

    srand(time(NULL));

    for(i = 0; i < size; i++)
        m.elements[i] = rand() / (float)RAND_MAX;
}

////////////////////////////////////////////////////////////////////////////////
// main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) 
{
    int width = 0, height = 0;
    FILE *f, *out;
    if(argc < 2)
    {
        printf("Argumente prea puține, trimiteți id-ul testului care trebuie rulat\n");
        return 0;
    }
    char name[100];
    sprintf(name, "./tests/test_%s.txt", argv[1]);
    f = fopen(name, "r");
    out = fopen("out.txt", "a");
    fscanf(f, "%d%d", &width, &height);
    Matrix M;//kernel de pe host
    Matrix N;//matrice inițială de pe host
    Matrix P;//rezultat fără memorie partajată calculat pe GPU
    Matrix PS;//rezultatul cu memorie partajată calculat pe GPU
    
    M = AllocateMatrix(KERNEL_SIZE, KERNEL_SIZE);
    N = AllocateMatrix(width, height);        
    P = AllocateMatrix(width, height);
    PS = AllocateMatrix(width, height);

    GenerateRandomMatrix(M);
    GenerateRandomMatrix(N);
    

    //test dummy, pentru a elimina acea anomalie, vezi README
    printf("Test dummy:\t");
    ConvolutionOnDeviceShared(M, N, PS);

    // M * N pe device
    ConvolutionOnDevice(M, N, P);
    
    // M * N pe device cu memorie partajată
    ConvolutionOnDeviceShared(M, N, PS);

    // calculează rezultatul pe CPU pentru comparație
    Matrix reference = AllocateMatrix(P.width, P.height);

    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);
    sdkStartTimer(&kernelTime);
    computeGold(reference.elements, M.elements, N.elements, N.height, N.width);
    sdkStopTimer(&kernelTime);
    printf ("Timp execuție seriala: %f ms\n", sdkGetTimerValue(&kernelTime));

    int i;
    int x, y; x = y  = 0;

    for(i = 0; i < PS.width * PS.height; i++)
    {
        if(!(PS.elements[i] > reference.elements[i] + 0.1 || PS.elements[i] < reference.elements[i] - 0.1)) x++;
        y++;
    }

    // verifică dacă rezultatul obținut pe device este cel așteptat
    int res = CompareMatrices(reference, P);
    printf("Test global %s\n", (1 == res) ? "PASSED" : "FAILED");
    fprintf(out, "Test global %s %s\n", argv[1], (1 == res) ? "PASSED" : "FAILED");

    // verifică dacă rezultatul obținut pe device cu memorie partajată este cel așteptat
  //  int ress = CompareMatrices(reference, PS);
    int ress = CompareMatrices(reference, PS);
    printf("Test shared %s\n", (1 == ress) ? "PASSED" : "FAILED");
    fprintf(out, "Test shared %s %s\n", argv[1], (1 == ress) ? "PASSED" : "FAILED");
   
    // Free matrices
    FreeMatrix(&M);
    FreeMatrix(&N);
    FreeMatrix(&P);
    FreeMatrix(&PS);

    fclose(f);
    fclose(out);
    return 0;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P)
{
    Matrix Md, Nd, Pd; //matricele corespunzătoare de pe device
    int size;
    //pentru măsurarea timpului de execuție în kernel
    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);

    //Aloc matricele de pe device
    Md.width = M.width;
    Md.height = M.height;
    Md.pitch = M.pitch;
    size = M.width * M.height * sizeof(float);
    hipMalloc( (void**) &Md.elements, size);
    hipMemcpy( Md.elements, M.elements, size, hipMemcpyHostToDevice);

    Nd.width = N.width;
    Nd.height = N.height;
    Nd.pitch = N.pitch;
    size = N.width * N.height * sizeof(float);
    hipMalloc( (void**) &Nd.elements, size);
    hipMemcpy( Nd.elements, N.elements, size, hipMemcpyHostToDevice);


    Pd.width = P.width;
    Pd.height = P.height;
    Pd.pitch = P.pitch;
    size = P.width * P.height * sizeof(float);
    hipMalloc( (void**) &Pd.elements, size);
    

    //Setez configurația de rulare a kernelului
    dim3 dimGrid(N.width/BLOCK_SIZE + 1, N.height/BLOCK_SIZE + 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // sqrt(BLOCK_SIZE)


    sdkStartTimer(&kernelTime);

    //Lansez în execuție kernelul    
    ConvolutionKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd);

    hipDeviceSynchronize();
    sdkStopTimer(&kernelTime);
    printf ("Timp execuție kernel: %f ms\n", sdkGetTimerValue(&kernelTime));

    //Copiez rezultatul pe host
    hipMemcpy( P.elements, Pd.elements, size, hipMemcpyDeviceToHost);

    //Eliberez memoria matricelor de pe device
    hipFree(Md.elements);
    hipFree(Nd.elements);
    hipFree(Pd.elements);
}


void ConvolutionOnDeviceShared(const Matrix M, const Matrix N, Matrix P)
{
    Matrix Md, Nd, Pd; //matricele corespunzătoare de pe device
    int size;

    //pentru măsurarea timpului de execuție în kernel
    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);
    
    //Aloc matricele de pe device
    Md.width = M.width;
    Md.height = M.height;
    Md.pitch = M.pitch;
    size = M.width * M.height * sizeof(float);
    hipMalloc( (void**) &Md.elements, size);
    hipMemcpy( Md.elements, M.elements, size, hipMemcpyHostToDevice);

    Nd.width = N.width;
    Nd.height = N.height;
    Nd.pitch = N.pitch;
    size = N.width * N.height * sizeof(float);
    hipMalloc( (void**) &Nd.elements, size);
    hipMemcpy( Nd.elements, N.elements, size, hipMemcpyHostToDevice);

    Pd.width = P.width;
    Pd.height = P.height;
    Pd.pitch = P.pitch;
    size = P.width * P.height * sizeof(float);
    hipMalloc( (void**) &Pd.elements, size);
    

   //Setez configurația de rulare a kernelului
    dim3 dimGrid(N.width/BLOCK_SIZE + 1, N.height/BLOCK_SIZE + 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // sqrt(BLOCK_SIZE)


    sdkStartTimer(&kernelTime);

    //Lansez în execuție kernelul    
    ConvolutionKernelShared<<<dimGrid, dimBlock>>>(Md, Nd, Pd);
       
    hipDeviceSynchronize();
    sdkStopTimer(&kernelTime);
    printf ("Timp execuție kernel cu memorie partajata: %f ms\n", sdkGetTimerValue(&kernelTime));
    
    //Copiez rezultatul pe host
    hipMemcpy( P.elements, Pd.elements, size, hipMemcpyDeviceToHost);

    //Eliberez memoria matricelor de pe device
    hipFree(Md.elements);
    hipFree(Nd.elements);
    hipFree(Pd.elements);
}


// Alocă matrice pe host de dimensiune height*width
Matrix AllocateMatrix(int width, int height)
{
    Matrix M;
    M.width = M.pitch = width;
    M.height = height;
    int size = M.width * M.height;    
    M.elements = (float*) malloc(size*sizeof(float));
    return M;
}    

// Eliberează o matrice de pe device
void FreeDeviceMatrix(Matrix* M)
{
    hipFree(M->elements);
    M->elements = NULL;
}

// Eliberează o matrice de pe host
void FreeMatrix(Matrix* M)
{
    free(M->elements);
    M->elements = NULL;
}